#include <iostream>
#include <hip/hip_runtime.h>


__device__ const double __NEGATIVE_INFINITY__ = -1.7976931348623157e+308;
__device__ const double __POSITIVE_INFINITY__ = +1.7976931348623157e+308;


__global__ void cuda_array_assign(
    double* array,
    int length,
    double value
) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < length) array[index] = value;
}


__global__ void copy_from_operands(
    double *dest,
    double *operands,
    int *arrCpy,
    int length,
    int numCpy
) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < numCpy*length){
        int i = index / length;
        int j = index % length;
        dest[index] = operands[arrCpy[i]*length + j];
    }
}


__global__ void update_temp_weight(
    double *temp_weight_new,
    double *temp_weight_old,
    double *operands,
    int *arrOpr,
    int length,
    int numOpr,
    bool isMul
) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < numOpr*length){
        int i = index / length;
        int j = index % length;
        if (isMul)
            temp_weight_new[index] = temp_weight_old[j] * operands[arrOpr[i]*length + j];
        else
            temp_weight_new[index] = temp_weight_old[j] / operands[arrOpr[i]*length + j];
    }
}


__global__ void update_last_weight(
    double *last_weight,
    double *curr_weight,
    double *temp_weight,
    int length,
    int numOpr,
    bool isAdd
) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < numOpr*length){
        int j = index % length;
        if (isAdd) last_weight[index] = curr_weight[j] + temp_weight[index];
        else last_weight[index] = curr_weight[j] - temp_weight[index];
    }
}


__global__ void update_last_weight_through_operands(
    double *last_weight,
    double *curr_weight,
    double *operands,
    int *arrOpr,
    int length,
    int numOpr,
    bool isAdd
) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < numOpr*length){
        int i = index / length;
        int j = index % length;
        if (isAdd) last_weight[index] = curr_weight[j] + operands[arrOpr[i]*length + j];
        else last_weight[index] = curr_weight[j] - operands[arrOpr[i]*length + j];
    }
}


__global__ void replace_nan_and_inf(
    double *array,
    int length,
    int numOpr
) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < numOpr*length){
        if (isnan(array[index]) || isinf(array[index]))
            array[index] = __NEGATIVE_INFINITY__;
    }
}


__device__ double max_of_array(
    double *array,
    int left,
    int right,
    double supremum
) {
    double max_ = __NEGATIVE_INFINITY__;
    for (int i=left; i<right; i++){
        if (array[i] < supremum && array[i] > max_) max_ = array[i];
    }
    return max_;
}


__device__ void top_n_of_array(
    double *array,
    int left,
    int right,
    double *result,
    int start,
    int n
) {
    double supremum = __POSITIVE_INFINITY__;
    for (int i=0; i<n; i++){
        supremum = max_of_array(array, left, right, supremum);
        result[start+i] = supremum;
    }
}


__global__ void fill_thresholds(
    double *weights,
    double *thresholds,
    int *INDEX,
    int index_length,
    int num_array,
    int length
) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int num_cycle = index_length - 2;
    if (index < num_array*num_cycle){
        int ix = index % num_cycle;
        int iy = index / num_cycle;
        top_n_of_array(weights + iy*length,
                       INDEX[ix+1], INDEX[ix+2],
                       thresholds + iy*5*num_cycle,
                       ix*5, 5);
    }
}


__device__ void _multi_invest_2(
    double *weight,
    double threshold,
    int t_idx,
    double *result,
    double INTEREST,
    int *INDEX,
    double *PROFIT,
    int *SYMBOL,
    int *BOOL_ARG,
    int index_size,
    int num_cycle
) {
    int reason = 0;
    double Geo2 = 0, Har2 = 0;
    int start, end, end2, count, k, sym, s, rs_idx;
    double temp, n;
    bool check;
    for (int i=index_size-3; i>0; i--){
        start = INDEX[i];
        end = INDEX[i+1];
        temp = 0;
        count = 0;
        check = false;
        if (!reason){
            end2 = INDEX[i+2];
            for (k=start; k<end; k++){
                if (weight[k] > threshold && BOOL_ARG[k]){
                    check = true;
                    sym = SYMBOL[k];
                    for (s=end; s<end2; s++){
                        if (SYMBOL[s] == sym){
                            if (weight[s] > threshold){
                                count++;
                                temp += PROFIT[k];
                            }
                            break;
                        }
                    }
                }
            }
        } else {
            for (k=start; k<end; k++){
                if (weight[k] > threshold && BOOL_ARG[k]){
                    check = true;
                    count++;
                    temp += PROFIT[k];
                }
            }
        }

        if (!count){
            Geo2 += log(INTEREST);
            Har2 += 1.0 / INTEREST;
            if (!check) reason = 1;
        } else {
            temp /= count;
            Geo2 += log(temp);
            Har2 += 1.0 / temp;
            reason = 0;
        }

        if (i <= num_cycle && t_idx+1 >= i){
            rs_idx = num_cycle - i;
            n = index_size - 2 - i;
            result[2*rs_idx] = exp(Geo2/n);
            result[2*rs_idx+1] = n / Har2;
        }
    }
}


__global__ void multi_invest_2(
    double *weights,
    double *thresholds,
    double *results,
    int num_array,
    int num_threshold,
    int length,
    int num_cycle,
    double INTEREST,
    int *INDEX,
    double *PROFIT,
    int *SYMBOL,
    int *BOOL_ARG,
    int index_size
) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < num_array*num_threshold){
        int ix = index % num_threshold;
        int iy = index / num_threshold;
        _multi_invest_2(
            weights + iy*length,
            thresholds[iy*num_threshold + ix],
            ix / 5,
            results + iy*num_threshold*num_cycle*2 + ix*num_cycle*2,
            INTEREST, INDEX, PROFIT, SYMBOL, BOOL_ARG, index_size, num_cycle
        );
    }
}


__global__ void find_finals(
    double *results,
    double *thresholds,
    double *finals,
    int num_array,
    int num_threshold,
    int num_cycle
) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < 2*num_array*num_cycle){
        int iz = index % 2;
        int ix = (index/2) % num_cycle;
        int iy = (index/2) / num_cycle;

        double *result = results + iy*num_threshold*num_cycle*2;
        double *threshold = thresholds + iy*num_threshold;
        double *final_ = finals + iy*num_cycle*4 + ix*4;

        final_[2*iz] = threshold[0];
        final_[2*iz + 1] = result[2*ix + iz];
        for (int i=1; i<num_threshold; i++){
            if (result[i*num_cycle*2 + 2*ix + iz] > final_[2*iz + 1]){
                final_[2*iz] = threshold[i];
                final_[2*iz + 1] = result[i*num_cycle*2 + 2*ix + iz];
            }
        }
    }
}
